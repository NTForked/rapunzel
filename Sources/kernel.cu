#include "hip/hip_runtime.h"
/*------------------------------------------------------------------------------
/                             Proiect de diplomă
/
/                            Pavel Andrei-Cristian
/                               341C1, ACS, UPB
/
/                                 CUDA Modules
/-------------------------------------------------------------------------------
*/

#define M_PI	3.14159265358979323846
#include "HeadersAndDefines.h"
#include "Hair.h"

__device__ float delta;
__device__ Vector3D *dev_copy;

// referenceVertex is base hair.
__device__ float collisionOffset( Vector3D vertex, Vector3D referenceVertex, Vector3D centerOfGravity ){
	return referenceVertex.Distance( centerOfGravity ) - vertex.Distance( centerOfGravity );
}

__device__ float collisionOffsetAfterFall( Vector3D vertex, Vector3D referenceVertex, Vector3D centerOfGravity, float fallSpeed ){
	return referenceVertex.Distance( centerOfGravity ) - ( vertex - Vector3D( 0.0f, fallSpeed, 0.0f ) ).Distance( centerOfGravity );
}

__device__ float collisionOffsetAfterWind( Vector3D vertex, Vector3D referenceVertex, Vector3D centerOfGravity, Vector3D windDirection, float windSpeed ){
	return referenceVertex.Distance( centerOfGravity ) - ( vertex + windDirection * windSpeed ).Distance( centerOfGravity );
}

__device__ void restoreFromCollision( Vector3D &vertex, Vector3D previousVertex, Vector3D referenceVertex, Vector3D centerOfGravity ){
	// Declarations.
	float collisionOffset_v;
	
	// Collision with mannequin.
	collisionOffset_v = collisionOffset( vertex, referenceVertex, centerOfGravity );
	if( collisionOffset_v > 0 ){
		vertex += ( vertex - centerOfGravity ).normalize() * ( collisionOffset_v + RESTORATION_OFFSET );
	}

	// If real inter-node hair length is surpasses than permitted length
	if( vertex.Distance( previousVertex ) > HAIR_STRAND_LENGTH * HAIR_ELASTICITY_FACTOR ){
		// Restore it to normal length.
		vertex = previousVertex + ( vertex - previousVertex ).normalize() * HAIR_STRAND_LENGTH;
	}
}

__device__ void fall( Vector3D &vertex, Vector3D copy_vertex, Vector3D referenceVertex, Vector3D upsizedCenterOfGravity, Vector3D downsizedCenterOfGravity, float fallSpeed ){
	// If it's standing up	
	if( vertex.y > referenceVertex.y ){
		// Get it away from the mannequin.
		vertex += ( copy_vertex - upsizedCenterOfGravity ).normalize() * fallSpeed;
	}
	// If it's hanging down
	else{
		// Bring it closer to the mannequin.
		vertex += ( downsizedCenterOfGravity - copy_vertex ).normalize() * fallSpeed;
	}
	vertex.y -= fallSpeed;
}

__device__ void negativeGravity( Vector3D &vertex, Vector3D copy_vertex, Vector3D referenceVertex, Vector3D upsizedCenterOfGravity, Vector3D downsizedCenterOfGravity, float fallSpeed ){
	// If it's hanging down
	if( vertex.y < referenceVertex.y ){
		// Get it away from the mannequin.
		vertex += ( copy_vertex - downsizedCenterOfGravity ).normalize() * fallSpeed;
	}
	// If it's standing up
	else{
		// Bring it closer to the mannequin.
		vertex += ( upsizedCenterOfGravity - copy_vertex ).normalize() * fallSpeed;
	}
	// Let it go up.
	vertex.y += fallSpeed;
}

__device__ void blowWind( Vector3D &vertex, Vector3D windDirection, float windSpeed, float t ){
	// Linear.
	// vertex += windDirection * windSpeed;

	// Sinusoidal.
	vertex += windDirection * abs( sin( t ) ) * windSpeed;

	// Euler. Not functional.
	// vertex = previousVertex + windDirection * windSpeed * t;

	// Verlet. Not functional.
	// vertex = ( previousVertex - previousPreviousVertex + windDirection * t * t ) * windSpeed;
}

__global__ void tiltForward( Vector3D* dev_vertices, unsigned int vertices_size, float _rotation ){
	// Declarations.
	unsigned int index;
	float cosine, sine, y, z;
	
	index = blockIdx.x * blockDim.x + threadIdx.x;
	if( index < vertices_size ){
		cosine = (float)cos( _rotation * M_PI / 180.0f );
		sine = (float)sin( _rotation * M_PI / 180.0f );
		y = dev_vertices[index].y;
		z = dev_vertices[index].z;
		dev_vertices[index].y = cosine	* y + sine		* z;
		dev_vertices[index].z = -sine	* y + cosine	* z;
	}
}

__global__ void tiltSideways( Vector3D* dev_vertices, unsigned int vertices_size, float _rotation ){
	// Declarations.
	unsigned int index;
	float cosine, sine, x, y;
	
	index = blockIdx.x * blockDim.x + threadIdx.x;
	if( index < vertices_size ){
		cosine = (float)cos( _rotation * M_PI / 180.0f );
		sine = (float)sin( _rotation * M_PI / 180.0f );
		x = dev_vertices[index].x;
		y = dev_vertices[index].y;
		dev_vertices[index].x = cosine	* x + sine		* y;
		dev_vertices[index].y = -sine	* x + cosine	* y;
	}
}

__global__ void kernel( Vector3D *dev_vertices, Vector3D *dev_copy, unsigned int vertices_size, unsigned int hairLength, unsigned int gravity, Vector3D centerOfGravity, Vector3D hairUpsizedCenterOfGravity, Vector3D hairDownsizedCenterOfGravity, Vector3D windDirection, bool wind, float fallSpeed, float windSpeed, float timeSinceLastSimulation ){
	
	// Declarations.
	unsigned int index, remainder;
	
	// Simulation.
	index = blockIdx.x * blockDim.x + threadIdx.x;
//	for( i = index * 32; i < index * 32 + 32; i += 1 )
	if( index < vertices_size ){
		remainder = index % ( hairLength + 1 );
		if( remainder != 0 ){
			if( gravity == NORMAL_GRAVITY )
				fall( dev_vertices[index], dev_copy[index], dev_copy[index - remainder], hairUpsizedCenterOfGravity, hairDownsizedCenterOfGravity, fallSpeed );
			else if( gravity == NEGATIVE_GRAVITY )
				negativeGravity( dev_vertices[index], dev_copy[index], dev_copy[index - remainder], hairUpsizedCenterOfGravity, hairDownsizedCenterOfGravity, fallSpeed );
			if( wind ){
//				delta += DELTA_INCREMENT;
				blowWind( dev_vertices[index], windDirection, windSpeed, timeSinceLastSimulation );
//				if( stuck( vertices[index], previousVertices[index] ) )
//					vertices[i] += ( vertices[index] - centerOfGravity ).normalize() * 0.01f;
			}
			restoreFromCollision( dev_vertices[index], dev_copy[index - 1], dev_copy[index - remainder], centerOfGravity );
		//	previousVertices[index] = vertices[index];
		}
	}
}

__global__ void testKernel( Vector3D *dev_vertices, unsigned int vertices_size, unsigned int hairLength ){
	
	// Declarations.
	unsigned int index, remainder;
	
	// Simulation.
	index = blockIdx.x * blockDim.x + threadIdx.x;
	if( index < vertices_size ){
		remainder = index % ( hairLength + 1 );
		if( remainder != 0 ){
			dev_vertices[index].y += 0.01f;
		}
	}
}

__global__ void helloKernel(){
//	printf( "Hello from kernel!\n" );
}

hipError_t tiltKernelWrapper( Vector3D *dev_vertices, unsigned int vertices_size, unsigned int tiltDirection ){
	
	// Declarations.
	hipError_t cudaStatus;
	dim3 gridSize, blockSize;
	unsigned int affinity;
	
	// Launch a kernel on the GPU with one thread for each element.
	affinity = 512;
	gridSize = dim3( 1 + vertices_size / affinity, 1, 1 );
	blockSize = dim3( affinity, 1, 1 );

	switch( tiltDirection ){
	case 1 << TILT_FORWARD:
		tiltForward<<< gridSize, blockSize >>>( dev_vertices, vertices_size, TILT_SPEED );
		break;
	case 1 << TILT_BACKWARDS:
		tiltForward<<< gridSize, blockSize >>>( dev_vertices, vertices_size, -TILT_SPEED );
		break;
	case 1 << TILT_LEFT:
		tiltSideways<<< gridSize, blockSize >>>( dev_vertices, vertices_size, TILT_SPEED );
		break;
	case 1 << TILT_RIGHT:
		tiltSideways<<< gridSize, blockSize >>>( dev_vertices, vertices_size, -TILT_SPEED );
		break;
	}
	cudaStatus = hipDeviceSynchronize();
	if( cudaStatus != hipSuccess )
		errorHandler( cudaStatus, "tiltKernelWrapper()/hipDeviceSynchronize() failed.", hipGetErrorString( cudaStatus ) );
	return hipSuccess;
}

// Helper function for using CUDA to compute stencil operations in parallel.
hipError_t kernelWrapper( Vector3D *dev_vertices, unsigned int vertices_size, unsigned int hairLength, unsigned int gravity, Vector3D centerOfGravity, Vector3D upsizedCenterOfGravity, Vector3D downsizedCenterOfGravity, Vector3D windDirection, bool wind, float fallSpeed, float windSpeed, float &time, float timeSinceLastSimulation ){
	
	// Declarations.
	hipError_t cudaStatus;
	dim3 gridSize, blockSize;
	hipEvent_t start, stop;
	unsigned int affinity;
	
	cudaStatus = hipEventCreate( &start );
	if( cudaStatus != hipSuccess )
		errorHandler( cudaStatus, "hipEventCreate( &start ) failed.", hipGetErrorString( cudaStatus ) );

	cudaStatus = hipEventCreate( &stop );
	if( cudaStatus != hipSuccess )
		errorHandler( cudaStatus, "hipEventCreate( &stop ) failed.", hipGetErrorString( cudaStatus ) );
	
	cudaStatus = hipFree( dev_copy );
	if( cudaStatus != hipSuccess )
		errorHandler( cudaStatus, "kernel()/hipFree() failed.", hipGetErrorString( cudaStatus ) );

	cudaStatus = hipMalloc( &dev_copy, vertices_size * sizeof( Vector3D ) );
	if( cudaStatus != hipSuccess )
		errorHandler( cudaStatus, "kernel()/hipMalloc() failed.", hipGetErrorString( cudaStatus ) );

	cudaStatus = hipEventRecord( start, 0 );
	if( cudaStatus != hipSuccess )
		errorHandler( cudaStatus, "hipEventRecord( start, 0 ) failed.", hipGetErrorString( cudaStatus ) );

	hipMemcpy( dev_copy, dev_vertices, vertices_size * sizeof( Vector3D ), hipMemcpyDeviceToDevice );
		
	// Launch a kernel on the GPU with one thread for each element.
	affinity = 512;
	gridSize = dim3( 1 + vertices_size / affinity, 1, 1 );
	blockSize = dim3( affinity, 1, 1 );
	
//	sqrt3_affinity = 8;
//	pow_3 = pow( vertices_size, 0.33333333333333333333333333333333333333 );
//	gridSize = dim3( pow_3 / sqrt3_affinity, pow_3 / sqrt3_affinity, pow_3 / sqrt3_affinity );
//	blockSize = dim3( sqrt3_affinity, sqrt3_affinity, sqrt3_affinity );

//	gridSize = dim3( 1, 1, 1 );
//	blockSize = dim3( 1, 1, 1 );

	//helloKernel<<< gridSize, blockSize >>>();
//	testKernel<<< gridSize, blockSize >>>( dev_vertices, vertices_size, hair.length );
	kernel<<< gridSize, blockSize >>>( dev_vertices, dev_copy, vertices_size, hairLength, gravity, centerOfGravity, upsizedCenterOfGravity, downsizedCenterOfGravity, windDirection, wind, fallSpeed, windSpeed, timeSinceLastSimulation );

	cudaStatus = hipDeviceSynchronize();
	if( cudaStatus != hipSuccess )
		errorHandler( cudaStatus, "kernelWrapper()/hipDeviceSynchronize() failed.", hipGetErrorString( cudaStatus ) );

	cudaStatus = hipEventRecord( stop, 0 );
	if( cudaStatus != hipSuccess )
		errorHandler( cudaStatus, "hipEventRecord( stop, 0 ) failed.", hipGetErrorString( cudaStatus ) );
	
	cudaStatus = hipEventSynchronize( stop );
	if( cudaStatus != hipSuccess )
		errorHandler( cudaStatus, "hipEventSynchronize()1 failed.", hipGetErrorString( cudaStatus ) );

	cudaStatus = hipEventElapsedTime( &time, start, stop );
	if( cudaStatus != hipSuccess )
		errorHandler( cudaStatus, "hipEventElapsedTime() failed.", hipGetErrorString( cudaStatus ) );
	
//	hipMemcpy( dev_vertices, dev_copy, vertices_size * sizeof( Vector3D ), hipMemcpyDeviceToDevice );

		
	return hipSuccess;
}
